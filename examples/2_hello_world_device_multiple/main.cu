
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
  printf("Hello from the device!\n");
}

int main(void) {

  printf("Hello from the host!\n");

  mykernel<<<4,10>>>();
  hipDeviceSynchronize();

  return 0;
}

