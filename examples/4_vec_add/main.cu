
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mykernel(float *a, float *b, float *c) {
  int i = blockIdx.x;

  c[i] = a[i] + b[i];
  printf("device a[%d] = %f\n", blockIdx.x, a[blockIdx.x]);
}

int resultNoGood(hipError_t res) {
  return res != hipSuccess;
}

int main(int argc, char **argv) {

  if (argc < 2) {
    printf("usage: %s N\n", argv[0]);
    return 1;
  }

  long N = atoi(argv[1]);

  // pointers to host arrays
  float *h_a, *h_b, *h_c;

  // pointers to device arrays
  float *d_a, *d_b, *d_c;

  // allocate host memory
  h_a = (float*)malloc(N * sizeof(float));
  h_b = (float*)malloc(N * sizeof(float));
  h_c = (float*)malloc(N * sizeof(float));

  // allocate device memory
  hipError_t res_a = hipMalloc((void **)&d_a, N * sizeof(float));
  hipError_t res_b = hipMalloc((void **)&d_b, N * sizeof(float));
  hipError_t res_c = hipMalloc((void **)&d_c, N * sizeof(float));

  if (resultNoGood(res_a)) { printf("failed to allocate device memory for d_a\n"); return 1; }
  if (resultNoGood(res_b)) { printf("failed to allocate device memory for d_b\n"); return 1; }
  if (resultNoGood(res_c)) { printf("failed to allocate device memory for d_c\n"); return 1; }
    

  // initialize the arrays
  for (int i=0; i<N; i++) {
    h_a[i] = i;
    h_b[i] = 100 + i;
    h_c[i] = 0;
  }

  // copy data from host arrays to device arrays
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, N * sizeof(float), hipMemcpyHostToDevice);

  for (int i=0; i<N; i++) {
    printf("host a[%d] = %f\n", i, h_a[i]);
  }

  // launch kernel and 
  mykernel<<<N,1>>>(d_a, d_b, d_c);
  hipDeviceSynchronize();

  // copy data from device arrays to host arrays
  hipMemcpy(h_a, d_a, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_b, d_b, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

  printf("\nafter copying back device->host:\n");
  for (int i=0; i<N; i++) {
    printf("host c[%d] = %f\n", i, h_c[i]);
  }

  return 0;
}

